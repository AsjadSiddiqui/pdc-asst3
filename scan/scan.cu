#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}  

__global__ void scan_kernel_upsweep(int N, int* result, int span) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x)*span*2;
    int spanplus1 = 2*span;
    if (idx >= 0 && (idx + spanplus1 - 1 < N)) {
        result[idx + spanplus1 - 1] += result[idx + span - 1];
    }
}

__global__ void scan_kernel_downsweep(int N, int* result, int span) {
    int idx = (blockIdx.x * blockDim.x + threadIdx.x)*span*2;
    int spanplus1 = 2*span;
    if (idx>=0 && (idx + spanplus1 - 1 < N)) {
        int t = result[idx+span-1];
        result[idx + span - 1] = result[idx + spanplus1 - 1];
        result[idx + spanplus1 - 1] += t;
    }
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep input
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    
    //up-sweep phase
    N = nextPow2(N);
    int n_threads=0, blocks=0;
    for(int span = 1; span <= N/2; span*=2) {
        n_threads = N / span;
        if(n_threads > THREADS_PER_BLOCK) {
            blocks = (n_threads+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
            n_threads = THREADS_PER_BLOCK;
        }
        else blocks = 1;
        scan_kernel_upsweep<<<blocks, n_threads>>>(N, result, span);
        hipDeviceSynchronize();
    }
    hipMemset((void*)(result+N-1), 0, 1* sizeof(int));
    //down-sweep phase
    for(int span = N/2; span >= 1; span /= 2) {
        n_threads = N / span;
        if(n_threads > THREADS_PER_BLOCK) {
            blocks = (n_threads+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
            n_threads = THREADS_PER_BLOCK;
        }
        else blocks = 1;
        scan_kernel_downsweep<<<blocks, n_threads>>>(N, result, span);
        hipDeviceSynchronize();
    }
}  


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);

    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < (end-inarray);i++){
        std::cout << resultarray[i] << std::endl;
    }
    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

__global__ void find_equals(int* device_input, int* bools, int length) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx>=0 && idx < length-1) {
        bools[idx] = device_input[idx+1]==device_input[idx]?1:0;
    }
}

__global__ void find_idx(int* bools, int length, int* device_output) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx>=0 && idx < length-1) {
        if(bools[idx+1]-bools[idx]==1) device_output[bools[idx]] = idx;
    }
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    int N = nextPow2(length), blocks = 0;
    int* bools = nullptr;
    hipMalloc((void**)&bools, N*sizeof(int));
    int n_threads = length-1;
    if(n_threads > THREADS_PER_BLOCK) {
        blocks = (n_threads+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
        n_threads = THREADS_PER_BLOCK;
    }
    else blocks = 1;
    find_equals<<<blocks, n_threads>>>(device_input, bools, length);
    exclusive_scan(bools, N, bools);
    find_idx<<<blocks, n_threads>>>(bools, length, device_output);
    int ans;
    hipMemcpy(&ans, bools+length-1,sizeof(int), hipMemcpyDeviceToHost);
    return ans;
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
